#include <iostream>
#include <vector>
#include <string>
#include <algorithm>
#include <fstream>
#include <cmath>
#include <chrono>

#include "hip/hip_runtime.h"


using std::cout;
using std::endl;

/* Store a 2D array as a row major 1D array */
template <class T>
class array2D {
	int wid, ht;
	std::vector<T> data; /* wid * ht elements */
public:
	array2D(int w, int h) :wid(w), ht(h), data(w*h) {}

	// Return array size
	inline int nx() const { return wid; }
	inline int ny() const { return ht; }

	// Manipulate array elements
	 T &operator() (int x, int y) { return data[y*wid + x]; }
	 T operator() (int x, int y) const { return data[y*wid + x]; }

	// Swap our data with this array
	void swap(array2D<T> &other) {
		std::swap(wid, other.wid);
		std::swap(ht, other.ht);
		std::swap(data, other.data);
	}
};

/* Dump a 2D array to a PPM file */
template <class T>
void write(const array2D<T> &arr, const char *name) {
	std::ofstream f(name, std::ios_base::binary);
	f << "P5\n"; // grayscale
	f << arr.nx() << " " << arr.ny() << "\n"; // dimensions
	f << "255\n"; // byte data
	for (int y = 0;y<arr.ny();y++)
		for (int x = 0;x<arr.nx();x++)
		{
			float v = arr(x, y)*255.99;
			unsigned char c = (unsigned char)v;
			if (v<0) c = 0;
			if (v>255) c = 255;
			f.write((char *)&c, 1);
		}
}
__global__ void blur(float cur[1000][1000], float next[1000][1000])
{
	int y = threadIdx.x+1;
	int x = blockIdx.x+1;
	//array2D<float> cur = *curr, next = *nex;
	//for (int y=1;y<cur.ny()-1;y++)
	//for (int x=1;x<cur.nx()-1;x++)
	for (int iter = 0;iter < 100;++iter)
	{
		next[x - 1][y - 1] = 0.25*(cur[x - 1][y] + cur[x + 1][y] + cur[x][y - 1] + cur[x][y + 1]);

		float temp;
		temp = next[x - 1][y - 1];
		next[x - 1][y - 1] = cur[x - 1][y - 1];
		cur[x - 1][y - 1] = temp;
	}
	
	
}
void I_pity_the_foo() {
	cout << "foo begin" << endl;
	const int w = 1000, h = 1000;
	cout << "foo creating 2DArrays" << endl;
	array2D<float> cur(w, h);
	array2D<float> next(w, h);

	cout << "foo creating 2D arrays" << endl;
	float curr[w][h], nex[w][h], dest[w][h];
	float gpu_curr[w][h], gpu_next[w][h];

	// Make initial conditions
	cout << "foo initializing arrays" << endl;
	for (int y = 0;y<cur.ny();y++)
		for (int x = 0;x<cur.nx();x++)
		{
			cur(x, y) = fmod(0.01*sqrt(x*x + y*y), 1.0);
			// subtle: need boundary conditions for next array
			next(x, y) = cur(x, y);
			curr[x][y] = cur(x, y);
			nex[x][y] = cur(x, y);
		}
	

	// Run a few iterations of blurring
	enum { nblur = 100 };

	cout << "foo allocating and copying arrays to gpu" << endl;
	hipMalloc((void**)&gpu_curr, w * h * sizeof(float));
	hipMalloc((void**)&gpu_next, w * h * sizeof(float));
	hipMemcpy(gpu_curr, curr, w * h * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(gpu_next, nex, w * h * sizeof(float), hipMemcpyHostToDevice);

	cout << "foo running blur on gpu" << endl;
	std::chrono::time_point<std::chrono::high_resolution_clock> start, end;
	start = std::chrono::high_resolution_clock::now();
	/* 	for (int blur=0;blur<nblur;blur++)
	{
	for (int y=1;y<cur.ny()-1;y++)
	for (int x=1;x<cur.nx()-1;x++)
	{
	next(x,y)=0.25*(cur(x-1,y)+cur(x+1,y)+cur(x,y-1)+cur(x,y+1));
	}
	cur.swap(next);
	} */
	blur<<<999, 999 >>>(gpu_curr, gpu_next);

	end = std::chrono::high_resolution_clock::now();;
	std::chrono::duration<double> elapsed = end - start;
	cout << "Performance: " << elapsed.count() / ((w - 2)*(h - 2)*nblur)*1.0e9 << " ns/pixel\n";

	hipDeviceSynchronize();

	cout << "foo finished bluring, copying data back from gpu" << endl;
	hipMemcpy(dest, gpu_curr, w*h*sizeof(float), hipMemcpyDeviceToHost);

	cout << "foo cleaning up gpu resources" << endl;
	hipFree(gpu_curr);
	hipFree(gpu_next);
	hipDeviceReset();
	
	cout << "foo writing blurred data back to 2DArray class" << endl;
	for (int y = 0;y<cur.ny();y++)
		for (int x = 0;x<cur.nx();x++)
		{
			cur(x, y) = dest[x][y];
		}
	// Dump final image (good for debugging)
	write(cur, "out.ppm");
	cout << "foo complete" << endl;
}


int main()
{
	cout << "pre foo" << endl;
	try {
		I_pity_the_foo();
	}
	catch (const std::exception & e)
	{
		cout << e.what() << endl;
	}
	cout << "post foo" << endl;
	return 0;
}