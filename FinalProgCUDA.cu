#include <iostream>
#include <vector>
#include <string>
#include <algorithm>
#include <fstream>
#include <cmath>
#include <chrono>

#include "hip/hip_runtime.h"


using std::cout;
using std::endl;

/* Store a 2D array as a row major 1D array */
template <class T>
class array2D {
	int wid, ht;
	std::vector<T> data; /* wid * ht elements */
public:
	array2D(int w, int h) :wid(w), ht(h), data(w*h) {}

	// Return array size
	inline int nx() const { return wid; }
	inline int ny() const { return ht; }

	// Manipulate array elements
	 T &operator() (int x, int y) { return data[y*wid + x]; }
	 T operator() (int x, int y) const { return data[y*wid + x]; }

	// Swap our data with this array
	void swap(array2D<T> &other) {
		std::swap(wid, other.wid);
		std::swap(ht, other.ht);
		std::swap(data, other.data);
	}
};

/* Dump a 2D array to a PPM file */
template <class T>
void write(const array2D<T> &arr, const char *name) {
	std::ofstream f(name, std::ios_base::binary);
	f << "P5\n"; // grayscale
	f << arr.nx() << " " << arr.ny() << "\n"; // dimensions
	f << "255\n"; // byte data
	for (int y = 0;y<arr.ny();y++)
		for (int x = 0;x<arr.nx();x++)
		{
			float v = arr(x, y)*255.99;
			unsigned char c = (unsigned char)v;
			if (v<0) c = 0;
			if (v>255) c = 255;
			f.write((char *)&c, 1);
		}
}
__global__ void blur(float *cur, float *next)
{
	int w =999;
	int y = threadIdx.x+1;
	int x = blockIdx.x+1;
	float temp;
	for (int iter = 0;iter < 100;++iter)
	{
		next[y*w+x] = 0.25*(cur[x - 1+w*y] + cur[x + 1+w*y] + cur[x+w*(y - 1)] + cur[x+w*(y + 1)]);

		temp = next[x+w*y];
		next[x+w*y] = cur[x+w*y];
		cur[x+w*y] = temp;
		
	}
	
	
}
void I_pity_the_foo() {
	//cout << "foo begin" << endl;
	const int w = 1000, h = 1000;
	//cout << "foo creating Array2Ds" << endl;
	array2D<float> cur(w, h);
	array2D<float> next(w, h);

	//cout << "foo creating 2D arrays" << endl;

	float host_curr[w*h], host_next[w*h];
	float * host_dest = new float[w*h];
	float *gp_curr = nullptr;
	float *gp_next = nullptr;

	// Make initial conditions
	//cout << "foo initializing arrays" << endl;
	for (int y = 0;y<cur.ny();y++)
		for (int x = 0;x<cur.nx();x++)
		{
			cur(x, y) = fmod(0.01*sqrt(x*x + y*y), 1.0);
			// subtle: need boundary conditions for next array
			next(x, y) = cur(x, y);
			host_curr[y*w+x]=cur(x,y);
			host_next[y*w+x]=cur(x,y);
		}
	

	// Run a few iterations of blurring
	enum { nblur = 100 };

	//cout << "foo allocating and copying arrays to gpu" << endl;


	//cout << "foo running blur on gpu" << endl;
	std::chrono::time_point<std::chrono::high_resolution_clock> start, end;
	start = std::chrono::high_resolution_clock::now();
	
	
	hipMalloc((void**)&gp_curr, w * h * sizeof(float));
	hipMalloc((void**)&gp_next, w * h * sizeof(float));
	hipMemcpy(gp_curr, host_curr, w * h * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(gp_next, host_next , w * h * sizeof(float), hipMemcpyHostToDevice);
	blur<<<999, 999 >>>(gp_curr, gp_next);
	hipDeviceSynchronize();
	hipMemcpy(host_dest, gp_curr, w*h*sizeof(float), hipMemcpyDeviceToHost);
	
	
	end = std::chrono::high_resolution_clock::now();;
	std::chrono::duration<double> elapsed = end - start;
	cout << "Performance: " << elapsed.count() / ((w - 2)*(h - 2)*nblur)*1.0e9 << " ns/pixel\n";

	

	//cout << "foo finished bluring, copying data back from gpu" << endl;


	//cout << "foo cleaning up gpu resources" << endl;
	hipFree(gp_curr);
	hipFree(gp_next);
	hipDeviceReset();
	
	
	//cout << "foo writing blurred data back to 2DArray class" << endl;
	for (int y = 0;y<cur.ny();y++)
		for (int x = 1;x<cur.nx()-1;x++)
		{
			cur(x, y) = host_dest[x+w*y];
		}
		
	//cout << "foo writing image output" << endl;	
	// Dump final image (good for debugging)
	write(cur, "out.ppm");

	delete[] host_dest;

	//cout << "foo complete" << endl;
}


int main()
{
	//cout << "pre foo" << endl;
	try {
		I_pity_the_foo();
	}
	catch (const std::exception & e)
	{
		cout << e.what() << endl;
	}
	//cout << "post foo" << endl;
	return 0;
}